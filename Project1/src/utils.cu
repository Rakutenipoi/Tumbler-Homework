#include "hip/hip_runtime.h"
#include "../include/utils.cuh"

void updateParticle_h(ParticleParameter* particleData, float* deltaTime) {
	ParticleParameter* particleData_d;
	float* deltaTime_d;

	// allocate devide memory
	hipMalloc((void**)&particleData_d, sizeof(ParticleParameter));
	hipMalloc((void**)&deltaTime_d, sizeof(float));

	// transfer the data from host memory to device memory
	hipMemcpy(particleData_d, particleData, sizeof(ParticleParameter), hipMemcpyHostToDevice);
	hipMemcpy(deltaTime_d, deltaTime, sizeof(float), hipMemcpyHostToDevice);

	updateParticle<<<4, 6>>>(particleData_d, deltaTime_d);

	// transfer the data from device memory to host memory
	hipMemcpy(particleData, particleData_d, sizeof(ParticleParameter), hipMemcpyDeviceToHost);

	// delete the device memory
	hipFree(particleData_d);
	hipFree(deltaTime_d);
}

__global__ void updateParticle(ParticleParameter * particleData, float* deltaTime) {
	vec3 position = particleData->position;
	vec3 direction = particleData->direction;
	vec3 acceleration = particleData->acceleration;
	float velocity = particleData->velocity;

	vec3 newVelocityVector = velocity * direction + acceleration * *deltaTime;
	vec3 newDirection = glm::normalize(newVelocityVector);
	float newVelocity = glm::length(newVelocityVector);

	position += newVelocityVector * *deltaTime;

	particleData->position = position;
	particleData->direction = newDirection;
	particleData->velocity = newVelocity;
}

