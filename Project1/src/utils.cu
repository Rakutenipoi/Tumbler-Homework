#include "hip/hip_runtime.h"
#include "../include/utils.cuh"

void updateParticle_h(ParticleParameter* particleData, float* deltaTime) {
	ParticleParameter* particleData_d;
	float* deltaTime_d;

	// allocate devide memory
	hipMalloc((void**)&particleData_d, sizeof(ParticleParameter));
	hipMalloc((void**)&deltaTime_d, sizeof(float));

	// transfer the data from host memory to device memory
	hipMemcpy(particleData_d, particleData, sizeof(ParticleParameter), hipMemcpyHostToDevice);
	hipMemcpy(deltaTime_d, deltaTime, sizeof(float), hipMemcpyHostToDevice);

	// updateParticle<<<256, 256>>>(particleData_d, deltaTime_d);

	// transfer the data from device memory to host memory
	hipMemcpy(particleData, particleData_d, sizeof(ParticleParameter), hipMemcpyDeviceToHost);

	// delete the device memory
	hipFree(particleData_d);
	hipFree(deltaTime_d);
}

__device__ void updateParticle(ParticleParameter * particleData, float* deltaTime) {
	vec3 position = particleData->position;
	vec3 direction = particleData->direction;
	vec3 acceleration = particleData->acceleration;
	float velocity = particleData->velocity;

	vec3 newVelocityVector = velocity * direction + acceleration * *deltaTime;
	vec3 newDirection = glm::normalize(newVelocityVector);
	float newVelocity = glm::length(newVelocityVector);

	position += *deltaTime * newVelocityVector;
	particleData->position = position;
	particleData->direction = newDirection;
	particleData->velocity = newVelocity;
}

void updateParticleSystem(Particle** particlesData, float* deltaTime, int numParticles) {
	Particle* particlePtr_d;
	Particle* particlePtr_h = new Particle[numParticles];
	float* deltaTime_d;
	int* numParticles_d;
	size_t particlesSize = numParticles * sizeof(Particle*);
	size_t particleSize = numParticles * sizeof(Particle);

	for (int i = 0; i < numParticles; i++) {
		hipMemcpy(&particlePtr_h[i], particlesData[i], sizeof(Particle), hipMemcpyHostToHost);
	}

	// allocate device memory
	//hipMalloc((void**)&particlesPtr_d, particlesSize);
	hipMalloc((void**)&particlePtr_d, particleSize);
	hipMalloc((void**)&deltaTime_d, sizeof(float));
	hipMalloc((void**)&numParticles_d, sizeof(int));

	// transfer the data from host to device
	hipMemcpy(particlePtr_d, particlePtr_h, particleSize, hipMemcpyHostToDevice);
	hipMemcpy(deltaTime_d, deltaTime, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(numParticles_d, &numParticles, sizeof(int), hipMemcpyHostToDevice);

	updateParticleSystem<<<256, 512>>>(particlePtr_d, deltaTime_d, numParticles_d);

	// transfer the data from device to host
	hipMemcpy(particlePtr_h, particlePtr_d, particleSize, hipMemcpyDeviceToHost);

	for (int i = 0; i < numParticles; i++) {
		hipMemcpy(particlesData[i], &particlePtr_h[i], sizeof(Particle), hipMemcpyHostToHost);
	}

	// delete device memory
	hipFree(particlePtr_d);
	hipFree(deltaTime_d);
	hipFree(numParticles_d);
	delete[] particlePtr_h;
}

__global__ void updateParticleSystem(Particle* particleData, float* deltaTime, int* numParticles) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < *numParticles) {
		Particle* particle = &particleData[idx];
		updateParticle(&particle->param, deltaTime);
	}
}

